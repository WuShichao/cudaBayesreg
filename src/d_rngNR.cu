#include "hip/hip_runtime.h"

#include <math.h>
#include "d_nr3.h"


__device__ struct __align__(16) Ran {
	Ullong u,v,w;
	 __device__ Ran(Ullong j) : v(4101842887655102017LL), w(1) {
	// Constructor. Call with any integer seed (except value of v above).
		u = j ^ v; int64();
		v = u; int64();
		w = v; int64();
	}
	
	 __device__ inline Ullong int64() {
		// Return 64-bit random integer. See text for explanation of method.
		u = u * 2862933555777941757LL + 7046029254386353087LL;
		v ^= v >> 17; v ^= v << 31; v ^= v >> 8;
		w = 4294957665U*(w & 0xffffffff) + (w >> 32);
		Ullong x = u ^ (u << 21); x ^= x >> 35; x ^= x << 4;
		return (x + v) ^ w;
	}
	
	 __device__ inline Doub doub() { return 5.42101086242752217E-20 * int64(); }
	// Return random double-precision ﬂoating value in the range 0. to 1.

	 __device__ inline Uint int32() { return (Uint)int64(); }
	// Return 32-bit random integer.

	// a random integer between 1 and n (inclusive)
	__device__ inline Int int1n(Uint n) { return(1 + int64() % (n-1)); }
};
	


struct __align__(16) Normaldev_BM : Ran {
	// Structure for normal deviates.
	Doub mu,sig;
	Doub storedval;
	 __device__ Normaldev_BM(Doub mmu, Doub ssig, Ullong i)
	: Ran(i), mu(mmu), sig(ssig), storedval(0.) {}
	// Constructor arguments are mu, sigma, and a random sequence seed.
	 __device__ Doub dev() {
		// Return a normal deviate.
		Doub v1,v2,rsq,fac;
		if (storedval == 0.) {
			do {
				v1=2.0*doub()-1.0;
				v2=2.0*doub()-1.0;
				rsq=v1*v1+v2*v2;
			} while (rsq >= 1.0 || rsq == 0.0);
			fac=sqrt(-2.0*log(rsq)/rsq);
		 	storedval = v1*fac;
			return mu + sig*v2*fac;
		} else {
				fac = storedval;
				storedval = 0.;
				return mu + sig*fac;
		}
	}
};


struct __align__(16) Gammadev : Normaldev_BM {
// Structure for gamma deviates.
	Doub alph, oalph, bet;
	Doub a1,a2;
	 __device__ Gammadev(Doub aalph, Doub bbet, Ullong i)
	: Normaldev_BM(0.,1.,i), alph(aalph), oalph(aalph), bet(bbet) {
	// Constructor arguments are ˛, ˇ , and a random sequence seed.
//		if (alph <= 0.) throw("bad alph in Gammadev");
		if (alph <= 0.) return;
		if (alph < 1.) alph += 1.;
		a1 = alph-1./3.;
		a2 = 1./sqrt(9.*a1);
	}
	 __device__ Doub dev() {
	// Return a gamma deviate by the method of Marsaglia and Tsang.
		Doub u,v,x;
		do {
			do {
				x = Normaldev_BM::dev();
				v = 1. + a2*x;
			} while (v <= 0.);
			v = v*v*v;
			u = doub();
		} while (u > 1. - 0.331*SQR(SQR(x)) &&
			log(u) > 0.5*SQR(x) + a1*(1.-v+log(v))); // Rarely evaluated.
		if (alph == oalph) return a1*v/bet;
		else { // Case where ˛ < 1, per Ripley.
			do u=doub(); while (u == 0.);
			return powf(u,1./oalph)*a1*v/bet;
		}
	}
};


__device__ void d_rnorm(Normaldev_BM* nd,  int n, float mu, float sig, Doub* res)
{
	for(int i=0; i<n; i++) 
		res[i] = nd->dev();
}
	
__device__ void d_rchisq(Gammadev* chi, int n, Doub* res)
{
	for(int i=0; i<n; i++) 
		res[i] = chi->dev();
}



