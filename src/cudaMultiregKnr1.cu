#include "hip/hip_runtime.h"
//$Id: cudaMultiregKnr1.cu,v 1.4 2010/05/15 14:39:39 afs Exp $
//
// !!! Processing in mycudamath functions assumes column order for compatibility with R
//

#include "d_rngBrent.cu"

namespace Kbrent {

using namespace Brent_rng;

__global__ void
cudaruniregNRK(float* d_betabar, float* tau, float* y, int nu, int nreg, int nobs, int m, int seed)
{
	const int ti = blockIdx.x * blockDim.x + threadIdx.x;
	if(ti >= nreg) return;
	const float df = nu+nobs; 
	const ulint seedti = (ulint)((seed >> 1)+ti);
	rngGamma drng(df / 2.0f, 0.5f, seedti);
//
	float* X = d_X;
	float* XpX = d_XpX;
	float* A = d_Abeta;
	float* ssq = d_ssq;
	const int mxm = m*m;
//
 	const float	sigmasq = tau[ti];
	//----------------------------
  // IR=backsolve(chol(XpX/sigmasq+A),diag(k))
	float IR[MDIM];
	{
		float tmp0[MDIM];
		for(int i=0; i < mxm; i++) 
			tmp0[i] = XpX[i] / sigmasq + A[i];
		mdgbacksolve(tmp0, &m,  IR);
	}
	//----------------------------
	// Xpy
	float Xpy[XDIM];
	float* yblock = &y[ti*nobs];

	mvtcrossp(X, yblock, Xpy, &nobs, &m);
	//----------------------------
	// btilde=crossprod(t(IR))%*%(Xpy/sigmasq+A%*%betabar)
	float* betabar = &d_betabar[ti*m];
	float btilde[XDIM];
	{
		float tmp1[XDIM];
		mvprod(A,betabar, tmp1, &m, &m);
		// (Xpy/sigmasq+A%*%betabar)
		for (int i=0; i<m; i++)
			tmp1[i] = Xpy[i] / sigmasq + tmp1[i]; 
		// crossprod(t(IR))
		float cIR[MDIM];
		mtcrossp(IR, IR, cIR, &m);
		mvprod(cIR, tmp1, btilde, &m, &m);
	}
	//----------------------------
  // beta = btilde + IR%*%rnorm(k)
	// Update betabar
	float beta[XDIM];
	{
		float tmp1[XDIM];
		for(int i=0; i < m; i++) tmp1[i]=drng.d_rnorm();
	 	// d_rnorm(&drng, m, tmp1);
		mvprod(IR, tmp1, beta, &m, &m);
	  for (int i=0; i < m; i++) 
			beta[i] = beta[i] + btilde[i]; 
	}
	//----------------------------
  // res=y-X%*%beta
  // s=t(res)%*%res
  // sigmasq=(nu*ssq + s)/rchisq(1,nu+n)
	float s;
	float resid[OBSDIM];
	mvprod(X, beta, resid, &nobs, &m);
	for(int i=0; i < nobs; i++) 
		resid[i] = yblock[i] - resid[i];
	vprod(resid, resid, &s, &nobs);
	float rchi;
	rchi = drng.d_rchisq();
	// d_rchisq(&drng, 1, &rchi);
	// d_rchisq(1, nu+nobs, 1234, &rchi);
	//----------------------------
	// Results
	tau[ti] = (nu*ssq[ti] + s)/rchi;
	//	__syncthreads();
	int ix=ti*m;
	for(int i=0; i < m; i++) {
		d_betabar[ix+i] = beta[i];
	}
}

}
